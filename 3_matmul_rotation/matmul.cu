#include "hip/hip_runtime.h"

#include <assert.h>
#include <hip/device_functions.h>
#include <stdio.h>
#include <exception>
#include <iomanip>
#include <iostream>
#include <random>
#include <string>
#include "hip/hip_runtime.h"
#include ""
#include "utils.cuh"

#define BLOCK_WIDTH 32

// CPU Matmul
void cpuMatMul(const float *A, const float *B, float *C, int M, int N, int K)
{
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < N; j++)
    {
      float sum = 0.0f;
      for (int k = 0; k < K; k++)
      {
        sum += A[i * N + k] * B[k * N + j];
      }
      C[i * N + j] = sum;
    }
  }
}

void cpuMatTranspose(const float *A, float *B, int M, int N)
{
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < N; j++)
    {
      B[j * M + i] = A[i * N + j];
    }
  }
}

bool verifyResults(const float *cpu_C, const float *gpu_C, int N,
                   float tolerance = 1e-5)
{
  for (int i = 0; i < N * N; i++)
  {
    if (fabs(cpu_C[i] - gpu_C[i]) > tolerance)
    {
      std::cout << "Mismatch at index " << i << " | CPU: " << cpu_C[i]
                << " vs GPU: " << gpu_C[i] << "\n";
      return false;
    }
  }
  std::cout << "GPU results match CPU results within tolerance " << tolerance
            << "\n";
  return true;
}

// Function to print a matrix
void printMatrix(const float *matrix, int M, int N)
{
  std::cout << "Matrix (" << M << "x" << N << "):\n";
  for (int i = 0; i < M; i++)
  {
    for (int j = 0; j < N; j++)
    {
      std::cout << matrix[i * N + j] << "\t";
    }
    std::cout << "\n";
  }
  std::cout << "-----------------------------------\n";
}

// Test kernel to fill and check 2D indexing
__global__ void fill2DIdx(float *A, float *B, float *C, int N)
{
  int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  int width_x = (gridDim.x * blockDim.x);
  int globalIdx = (row * width_x) + col;

  C[globalIdx] = globalIdx;
}

// Matmul - Global memory Access
__global__ void matmulKernelNaive(float *A, float *B, float *C, int M, int N,
                                  int K)
{
  int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  // global idx for Matrix Dimension, not thread dimension
  int globalIdx = (row * N) + col;

  if (row < M && col < N)
  {
    float value = 0.0f;
    for (int k = 0; k < K; k++)
    {
      value += (A[row * N + k] * B[k * N + col]);
    }
    C[globalIdx] = value;
  }
}

// Matmul - Shared memory Access
__global__ void matmulKernelShared(float *A, float *B, float *C, int M, int N,
                                   int K)
{
  // Shared Memory for tiles
  __shared__ float As[BLOCK_WIDTH][BLOCK_WIDTH];
  __shared__ float Bs[BLOCK_WIDTH][BLOCK_WIDTH];

  int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  int globalIdx =
      (row * N) + col; // global idx for Matrix Dimension, not thread dimension
  float value = 0.0f;

  // loop through tiles
  int num_tiles = (K + BLOCK_WIDTH - 1) / BLOCK_WIDTH;
  for (unsigned int tile = 0; tile < num_tiles; tile++)
  {
    // Load into Shared Memory
    if (row < M && (tile * BLOCK_WIDTH + threadIdx.x) < K)
    {
      // tiling in x_direction
      //                          row  * width       + col
      int tile_idx_A = row * K + (tile * BLOCK_WIDTH + threadIdx.x);
      As[threadIdx.y][threadIdx.x] = A[tile_idx_A];
    }
    if ((tile * BLOCK_WIDTH + threadIdx.y) < K && col < N)
    {
      // tiling in y_direction
      //                row  * width       + col
      int tile_idx_B = (tile * BLOCK_WIDTH + threadIdx.y) * N + col;
      Bs[threadIdx.y][threadIdx.x] = B[tile_idx_B];
    }

    __syncthreads();

    // Compute from shared memory
    for (int k = 0; k < BLOCK_WIDTH; k++)
    {
      value += (As[threadIdx.y][k] * Bs[k][threadIdx.x]);
    }

    __syncthreads();
  }

  C[globalIdx] = value;
}

// Transpose - Shared Memory
__global__ void matTransposeNaive(float *A, float *B, int M, int N)
{
  __shared__ float tile[BLOCK_WIDTH][BLOCK_WIDTH];

  int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  int globalIdx =
      (row * N) + col; // global idx for Matrix Dimension, not thread dimension

  // Load data into shared memory
  if (col < N && row < M)
  {
    tile[threadIdx.y][threadIdx.x] = A[globalIdx];
  }

  __syncthreads();

  // Write data back to global memory
  col = blockIdx.y * blockDim.x + threadIdx.x;
  row = blockIdx.x * blockDim.y + threadIdx.y;
  globalIdx = (row * N) + col;

  if (col < M && row < N)
  {
    B[globalIdx] = tile[threadIdx.x][threadIdx.y];
  }
}

// Transpose - Shared Memory Padded
__global__ void matTransposePadded(float *A, float *B, int M, int N)
{
  __shared__ float tile[BLOCK_WIDTH][BLOCK_WIDTH + 1];

  int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  int globalIdx =
      (row * N) + col; // global idx for Matrix Dimension, not thread dimension

  // Load data into shared memory
  if (col < N && row < M)
  {
    tile[threadIdx.y][threadIdx.x] = A[globalIdx];
  }

  __syncthreads();

  // Write data back to global memory
  col = blockIdx.y * blockDim.x + threadIdx.x;
  row = blockIdx.x * blockDim.y + threadIdx.y;
  globalIdx = (row * N) + col;

  if (col < M && row < N)
  {
    B[globalIdx] = tile[threadIdx.x][threadIdx.y];
  }
}

void printMemoryRequirements(unsigned long M, unsigned long N,
                             unsigned long K)
{
  const int mat_elt_count = (M * K) + (K * N) + (M * N); // square matrix A+B+C
  long double array_size_bytes = mat_elt_count * sizeof(float);
  long double array_size_gbytes = array_size_bytes / (1024 * 1024 * 1024);
  long double array_size_mbytes = array_size_bytes / (1024 * 1024);

  std::cout << "---Memory Requirements---" << std::endl;
  std::cout << "Matrix Dimensions=>  A:" << M * K << " B:" << K * N
            << " C:" << M * N << std::endl;
  std::cout << "Total Memory Size: " << array_size_gbytes << "GB, "
            << array_size_mbytes << "MB" << std::endl;
}

int main()
{
  // Choose GPU
  CUDA_CALL(hipSetDevice(0));

  try
  {
    enum Options
    {
      MATMUL_NAIVE,
      MATMUL_SHARED,
      TRANSPOSE_NAIVE,
      TRANSPOSE_PADDED
    };
    Options option = TRANSPOSE_PADDED;

    // A->MxK, B->KxN, C->MxN
    const int M = 2048; // Rows - A
    const int N = 2048; // Cols - B
    const int K = 2048; // Cols-A,Rows-B
    int elts_A = M * K;
    int elts_B = K * N;
    int elts_C = M * N;
    int mem_size_A = elts_A * sizeof(float);
    int mem_size_B = elts_B * sizeof(float);
    int mem_size_C = elts_C * sizeof(float);

    printMemoryRequirements(M, N, K);

    // Allocate host memory
    float *h_A, *h_B, *h_C, *h_C_cpu;
    h_A = (float *)malloc(mem_size_A);
    h_B = (float *)malloc(mem_size_B);
    h_C = (float *)malloc(mem_size_C);
    h_C_cpu = (float *)malloc(mem_size_C);

    // Initialize Host Data
    std::fill(h_A, h_A + elts_A, 2);
    std::fill(h_B, h_B + elts_B, 2);
    std::fill(h_C, h_C + elts_C, 0);
    std::fill(h_C_cpu, h_C_cpu + elts_C, 0);

    /*printMatrix(h_A,N);
        printMatrix(h_B, N);*/
    // CPU implementation
    // cpuMatMul(h_A, h_B, h_C_cpu, M, N, K);
    // cpuMatTranspose(h_A, h_C_cpu, M, K);

    // Device Memory
    CudaMemory<float> d_A(mem_size_A), d_B(mem_size_B), d_C(mem_size_C);

    // Copy data to device
    CUDA_CALL(hipMemcpy(d_A.get(), h_A, mem_size_A, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_B.get(), h_B, mem_size_B, hipMemcpyHostToDevice));

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH); // tile size
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x,
                 (M + dimBlock.y - 1) / dimBlock.y);

    // printKernelConfig(dimGrid, dimBlock);

    // Launch the matrix multiplication kernel
    switch (option)
    {
    case MATMUL_NAIVE:
      matmulKernelNaive<<<dimGrid, dimBlock>>>(d_A.get(), d_B.get(),
                                               d_C.get(), M, N, K);
      break;

    case MATMUL_SHARED:
      matmulKernelShared<<<dimGrid, dimBlock>>>(d_A.get(), d_B.get(),
                                                d_C.get(), M, N, K);
      break;

    case TRANSPOSE_NAIVE:
      matTransposeNaive<<<dimGrid, dimBlock>>>(d_A.get(), d_C.get(), M, K);
      break;

    case TRANSPOSE_PADDED:
      matTransposePadded<<<dimGrid, dimBlock>>>(d_A.get(), d_C.get(), M, K);
      break;
    default:
      break;
    }

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CALL(hipMemcpy(h_C, d_C.get(), mem_size_C, hipMemcpyDeviceToHost));

    // Verify GPU results
    // verifyResults(h_C_cpu, h_C, N);
    // printMatrix(h_C, M,N);
    //
    // hipDeviceReset - for profiling
    CUDA_CALL(hipDeviceReset());

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
  }
  catch (std::exception &e)
  {
    fprintf(stderr, "Exception: %s\n", e.what());
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}
