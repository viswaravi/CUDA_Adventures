﻿#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#define CHECK_CUDA(call)                                                   \
  {                                                                        \
    hipError_t err = call;                                                \
    if (err != hipSuccess) {                                              \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
      exit(1);                                                             \
    }                                                                      \
  }
  
#define CHECK_CUFFT(call)                               \
  {                                                     \
    hipfftResult err = call;                             \
    if (err != HIPFFT_SUCCESS) {                         \
      std::cerr << "cuFFT error: " << err << std::endl; \
      exit(1);                                          \
    }                                                   \
  }

__global__ void complexPointwiseMul(hipfftComplex* a, hipfftComplex* b,
                                    int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    float real = a[idx].x * b[idx].x - a[idx].y * b[idx].y;  // Real part
    float imag = a[idx].x * b[idx].y + a[idx].y * b[idx].x;  // Imaginary part
    a[idx].x = real;
    a[idx].y = imag;
  }
}

int main() {
  // Input and kernel
  float h_x[] = {1.0f, 2.0f, 3.0f, 4.0f};  // Input signal
  float h_h[] = {1.0f, -1.0f};             // Kernel
  int N = 4;                               // Input length
  int M = 2;                               // Kernel length
  int padded_size = 8;                     // Next power of 2 >= N + M - 1

  // Allocate host padded arrays
  float* h_x_padded = new float[padded_size]();
  float* h_h_padded = new float[padded_size]();
  for (int i = 0; i < N; i++)
    h_x_padded[i] = h_x[i];
  for (int i = 0; i < M; i++)
    h_h_padded[i] = h_h[i];

  // Allocate device memory
  float *d_x, *d_h;
  hipfftComplex *d_X, *d_H;
  CHECK_CUDA(hipMalloc(&d_x, padded_size * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_h, padded_size * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_X, padded_size * sizeof(hipfftComplex)));
  CHECK_CUDA(hipMalloc(&d_H, padded_size * sizeof(hipfftComplex)));

  // Copy data to device
  CHECK_CUDA(hipMemcpy(d_x, h_x_padded, padded_size * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_h, h_h_padded, padded_size * sizeof(float),
                        hipMemcpyHostToDevice));

  // Create cuFFT plans
  hipfftHandle plan_r2c, plan_c2r;
  CHECK_CUFFT(hipfftPlan1d(&plan_r2c, padded_size, HIPFFT_R2C, 1));
  CHECK_CUFFT(hipfftPlan1d(&plan_c2r, padded_size, HIPFFT_C2R, 1));

  // Execute FFT (real to complex)
  CHECK_CUFFT(hipfftExecR2C(plan_r2c, d_x, d_X));
  CHECK_CUFFT(hipfftExecR2C(plan_r2c, d_h, d_H));

  // Pointwise multiplication
  int threadsPerBlock = 256;
  int blocks = (padded_size + threadsPerBlock - 1) / threadsPerBlock;
  complexPointwiseMul<<<blocks, threadsPerBlock>>>(d_X, d_H, padded_size);
  CHECK_CUDA(hipDeviceSynchronize());

  // Inverse FFT (complex to real)
  float* d_y = d_x;  // Reuse d_x for output
  CHECK_CUFFT(hipfftExecC2R(plan_c2r, d_X, d_y));

  // Copy result back
  float* h_y = new float[padded_size];
  CHECK_CUDA(hipMemcpy(h_y, d_y, padded_size * sizeof(float),
                        hipMemcpyDeviceToHost));

  // Normalize (cuFFT doesn't scale by 1/N)
  int out_size = N + M - 1;
  for (int i = 0; i < out_size; i++) {
    h_y[i] /= padded_size;
    std::cout << "y[" << i << "] = " << h_y[i] << std::endl;
  }

  // Cleanup
  CHECK_CUFFT(hipfftDestroy(plan_r2c));
  CHECK_CUFFT(hipfftDestroy(plan_c2r));
  CHECK_CUDA(hipFree(d_x));
  CHECK_CUDA(hipFree(d_h));
  CHECK_CUDA(hipFree(d_X));
  CHECK_CUDA(hipFree(d_H));
  delete[] h_x_padded;
  delete[] h_h_padded;
  delete[] h_y;

  return 0;
}